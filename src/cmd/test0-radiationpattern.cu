#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <thrust/host_vector.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/counting_iterator.h>

#include <sinr/radiationpattern.cuh>
#include <sinr/util.h>

using namespace std;

typedef double Real;


/** This test program plots radiation patterns and their total radiated power (TRP).  The total radiated power should be
 * at or very close to 1.
 */
int main(int argc __attribute__((unused)), char** argv __attribute__((unused))) {
  
  /* TRP parameters */
  Real trp;
  Real theta_a = 0.0;
  Real theta_b = 2.0*M_PI;
  Real N = 10000;
  thrust::counting_iterator<Real> cbeg(0);
  thrust::counting_iterator<Real> cend(N);
 
  /* plotting variables */
  thrust::host_vector<Real> rad(N);
  thrust::host_vector<Real> the(N);
  vector<Real> theta;
  vector<Real> radius;
 
  thrust::transform(thrust::make_transform_iterator(cbeg, Interpolate<Real>(0, 360, N)),
                    thrust::make_transform_iterator(cend, Interpolate<Real>(0, 360, N)),
                    the.begin(),
                    thrust::identity<Real>());
  theta = vector<Real>(the.begin(), the.end());



  /* ellipse parameters */
  Real ecc  = 0.9;
  Real norm = RadPatternEllipse<Real>::normalize(ecc);
 
  thrust::transform(thrust::make_transform_iterator(cbeg, Interpolate<Real>(theta_a, theta_b, N)),
                    thrust::make_transform_iterator(cend, Interpolate<Real>(theta_a, theta_b, N)),
                    rad.begin(),
                    RadPatternEllipse<Real>(ecc, norm));
  radius = vector<Real>(rad.begin(), rad.end());
  

  /* integrate radiation pattern from [0,2*pi] and make sure area is near 1. */
  trp = totalRadiatedPower<Real>(RadPatternEllipse<Real>(ecc, norm));
  cout<<"Ellipse Area: "<<trp<<endl;



  /* rose petal parameters */
  Real width = M_PI/4.0;
  Real power = 1;
 
  thrust::transform(thrust::make_transform_iterator(cbeg, Interpolate<Real>(theta_a, theta_b, N)),
                    thrust::make_transform_iterator(cend, Interpolate<Real>(theta_a, theta_b, N)),
                    rad.begin(),
                    RadPatternRosePetal<Real>(width, power));
  radius = vector<Real>(rad.begin(), rad.end());
 
  
  /* integrate radiation pattern from [0,2*pi] and make sure area is near 1. */
  trp = totalRadiatedPower<Real>(RadPatternRosePetal<Real>(width, power));
  cout<<"Rose Petal Area: "<<trp<<endl;

  return 0;
}
